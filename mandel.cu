
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <getopt.h>
#include <string.h>
#include <math.h>

#define MAX_MANDELBROT_ITERATIONS (80)
#define DEFAULT_OUTPUT "mandelbrot.bmp"
#define THREADS_PER_BLOCK (1024/2)

unsigned int imageWidth = 1920;
unsigned int imageHeight = 1080;
unsigned int rowPadding;

const struct option options[] = {
        {"help", 0, NULL, 1},
        {"xleft", 1, NULL, 2},
        {"xright", 1, NULL, 3},
        {"ylower", 1, NULL, 4},
        {"yupper", 1, NULL, 5},
        {"width", 1, NULL, 6},
        {"height", 1, NULL, 7},
        {"output", 1, NULL, 8},
        {0, 0, 0, 0}
};
    
const char helpMessage[] = "Usage: %s [args]\n\
Mandelbrot-BMP generates a BMP image of a specified location in the Mandelbrot set.\n\n\
Possible arguments:\n\
    --help\n\
    --xleft [value]    Set the leftmost x value to render    (double)\n\
    --xright [value]   Set the rightmost x value to render   (double)\n\
    --ylower [value]   Set the lowest y value to render      (double)\n\
    --yupper [value]   Set the highest y value to render     (double)\n\
    --width [value]    Set the width of the output image     (int)\n\
    --height [value]   Set the height of the output image    (int)\n\
    --output [value]   Set the location of the output image  (string)\n\n\
If only 3 boundary values are specified, the other can be inferred from the image aspect ratio.\n";

struct __attribute__((__packed__)) BITMAPFILEHEADER {
  uint16_t bfType;
  uint32_t bfSize;
  uint16_t bfReserved1;
  uint16_t bfReserved2;
  uint32_t bfOffBits;
};

struct __attribute__((__packed__)) BITMAPINFOHEADER {
  uint32_t biSize;
  uint32_t biWidth;
  uint32_t biHeight;
  uint16_t biPlanes;
  uint16_t biBitCount;
  uint32_t biCompression;
  uint32_t biSizeImage;
  uint32_t biXPelsPerMeter;
  uint32_t biYPelsPerMeter;
  uint32_t biClrUsed;
  uint32_t biClrImportant;
};

__device__ int inMandelbrotSet(double x, double y)
{
    double real = x;
    double imaginary = y;

    for (int i = 0; i < MAX_MANDELBROT_ITERATIONS; i++) {
        double real2 = real * real;
        double imaginary2 = imaginary * imaginary;
        imaginary = real * imaginary * 2 + y;
        real = real2 - imaginary2 + x;
        if (real2 + imaginary2 > 4)
            return i;
    }
    return MAX_MANDELBROT_ITERATIONS;
}

__global__ void renderPixel(unsigned char* bufStart, int width, int height, double xStart, double xRange, double yStart, double yRange, int padding, int iters)
{
    for(int i = (blockIdx.x * blockDim.x + threadIdx.x) * iters; i < (blockIdx.x * blockDim.x + threadIdx.x) * iters + iters; i++) {
        if(i >= (width * height))
            return;
        unsigned int x = i % width;
        unsigned int y = i / width;
        
        unsigned char gradientR = (unsigned char)((float)x / ((float)(width - 1) / 127.5) + (float)y / ((float)(height - 1) / 127.5));
        unsigned char gradientG = (unsigned char)((float)(width - x - 1) / ((float)(width - 1) / 127.5) + (float)(height - y - 1) / ((float)(height - 1) / 127.5));
        unsigned char gradientB = (unsigned char)((float)x / ((float)(width - 1) / 127.5) + (float)(height - y - 1) / ((float)(height - 1) / 127.5));

        unsigned int mandel = inMandelbrotSet(((double)x / (double)width) * xRange + xStart,
           ((double)y / (double)height) * yRange + yStart);

        unsigned int baseIndex = y * padding + y * width * 3 + x * 3;
        if (mandel > 4) {
            float factor = (float)(MAX_MANDELBROT_ITERATIONS - (mandel - 5)) / MAX_MANDELBROT_ITERATIONS;
            bufStart[baseIndex] = factor * gradientB;
            bufStart[baseIndex + 1] = factor * gradientG;
            bufStart[baseIndex + 2] = factor * gradientR;
        } else {
            bufStart[baseIndex] = gradientB;
            bufStart[baseIndex + 1] = gradientG;
            bufStart[baseIndex + 2] = gradientR;
        }
    }
}

void setDefaultCorners(double corners[])
{
    printf("Warning: Falling back to default Mandelbrot corners.\n");
    corners[0] = -2.4;
    corners[1] = 1.4;
    corners[2] = -0.5 * (corners[1] - corners[0]) * ((double)imageHeight / (double)imageWidth);
    corners[3] = -1 * corners[2];
}

/*
* corners should be left x, right x, lower y, upper y
* returns 1 if the program should proceed
* REMEMBER TO FREE *fileName!
*/
int getArgs(int argc, char** argv, double corners[4], char** fileName)
{
    for(int i = 0; i < 4; i++)
        corners[i] = NAN;
    *fileName = NULL;
    
    int res;
    int tmpImageHeight;
    int tmpImageWidth;
    while((res = getopt_long_only(argc, argv, "", options, NULL)) != -1) {
        switch(res) {
            case '?':
                printf("Run '%s --help' for more information\n", argv[0]);
                break;
            case 1:
                printf(helpMessage, argv[0]);
                return 0;
                break;
            case 2:
                corners[0] = strtod(optarg, NULL);
                break;
            case 3:
                corners[1] = strtod(optarg, NULL);
                break;
            case 4:
                corners[2] = strtod(optarg, NULL);
                break;
            case 5:
                corners[3] = strtod(optarg, NULL);
                break;
            case 6:
                tmpImageWidth = atoi(optarg);
                if(tmpImageWidth > 0 && tmpImageWidth < 20000)
                    imageWidth = tmpImageWidth;
                else
                    printf("Warning: Image width is not between 0 and 20000 pixels! Falling back to default.\n");
                break;
            case 7:
                tmpImageHeight = atoi(optarg);
                if(tmpImageHeight > 0 && tmpImageHeight < 20000)
                    imageHeight = tmpImageHeight;
                else
                    printf("Warning: Image height is not between 0 and 20000 pixels! Falling back to default.\n");
                break;
            case 8:
                if(*fileName != NULL)
                    free(*fileName);
                *fileName = strdup(optarg);
                break;
        }
    }
    
    if(!isnan(corners[0]) && !isnan(corners[1]) && (isnan(corners[2]) != isnan(corners[3]))) {
         double otherRange = (corners[1] - corners[0]) * ((double)imageHeight / (double)imageWidth);
         if(!isnan(corners[2]))
            corners[3] = corners[2] + otherRange;
         else if(!isnan(corners[3]))
            corners[2] = corners[3] - otherRange;
    } else if(!isnan(corners[2]) && !isnan(corners[3]) && (isnan(corners[0]) != isnan(corners[1]))) {
        double otherRange = (corners[3] - corners[2]) * ((double)imageWidth / (double)imageHeight);
         if(!isnan(corners[0]))
            corners[1] = corners[0] + otherRange;
         else if(!isnan(corners[1]))
            corners[0] = corners[1] - otherRange;
    } else if(!(!isnan(corners[0]) && !isnan(corners[1]) && !isnan(corners[2]) && !isnan(corners[3]))) {
        setDefaultCorners(corners);
    }
    
    if(*fileName == NULL)
        *fileName = strdup(DEFAULT_OUTPUT);
    
    return 1;
}

void gpuAssert(hipError_t retVal)
{
    if(retVal != hipSuccess) {
        printf("CUDA assertion failed: %s\n", hipGetErrorString(retVal));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char** argv)
{
    char* fileName;
    double corners[4];
    if(!getArgs(argc, argv, corners, &fileName))
        return 0;
    rowPadding = (4 - (imageWidth % 4)) % 4;
    
    unsigned int dataSize = imageWidth * 3 * imageHeight + rowPadding * imageHeight;
    struct BITMAPFILEHEADER bmpHeader = {0x4D42, 54 + dataSize, 0, 0, 54}; /* data obtained from MSDN and Wikipedia */
    struct BITMAPINFOHEADER bmpInfo = { sizeof(struct BITMAPINFOHEADER), imageWidth, imageHeight, 1, 24, 0, dataSize, 2835, 2835, 0, 0 };

    FILE* bmpOut = fopen(fileName, "wb");
    if(bmpOut == NULL) {
        perror("Error opening file");
        return -1;
    }

    fwrite(&bmpHeader, 1, sizeof(struct BITMAPFILEHEADER), bmpOut);
    fwrite(&bmpInfo, 1, sizeof(struct BITMAPINFOHEADER), bmpOut);

    unsigned char* gpuImg;
    gpuAssert(hipMalloc(&gpuImg, dataSize));
    renderPixel<<<(imageWidth * imageHeight) / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK>>>(gpuImg, imageWidth, imageHeight, corners[0], corners[1] - corners[0], corners[2], corners[3] - corners[2], rowPadding, 2);

    unsigned char* img = (unsigned char*) malloc(dataSize);
    gpuAssert(hipMemcpy(img, gpuImg, dataSize, hipMemcpyDeviceToHost));
    fwrite(img, 1, dataSize, bmpOut);
    gpuAssert(hipFree(gpuImg));
    free(img);
    free(fileName);
    fclose(bmpOut);
    return EXIT_SUCCESS;
}
